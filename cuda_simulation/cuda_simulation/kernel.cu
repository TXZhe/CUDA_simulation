#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <vector>

#define USRNUM 50

__global__ void SIContagionProcessOnGPU(const int *head, const int *len, const int *edges, int *Inum, int N)
{
	int i = threadIdx.x;
	int countI = 1;
	int step = 0;
	int *infected = new int[N];
	for (int i = 0; i < N; i++)
	{
		infected[i] = -1;
	}
	infected[0] = 0;

	Inum[step] = 1;
	while (countI < N)
	{
		step++;
		for (int i = 0; i < N; i++)
		{
			if (infected[i] < step && infected[i]!=-1)
			{
				for (int j = 0; j < len[i]; j++)
				{
					if (infected[edges[head[i] + j]] == -1)
					{
						infected[edges[head[i] + j]] = step;
						countI++;
					}
				}
			}
		}
		Inum[step] = countI;
	}
}

int main()
{
	std::vector< std::vector<int> > h_adjlist;
	h_adjlist.resize(USRNUM);

	printf("Start to read the network...\n");
	FILE *p1;
	p1 = fopen("F:\\private\\17210720150\\cuda_simulation\\data\\WS_50_4_0.3.txt", "r");
	int edge_num = 0;
	while (!feof(p1))
	{
		int a, b;
		fscanf(p1, "%d %d", &a, &b);
		h_adjlist[a].push_back(b);
		h_adjlist[b].push_back(a);
		edge_num += 2;
	}

	// change adj_list into head matrix & len matrix & edges matrix
	int *h_head, *h_len, *h_edges, *h_Inum;
	h_head = (int *)malloc(USRNUM * sizeof(int));
	h_len = (int *)malloc(USRNUM * sizeof(int));
	h_edges = (int *)malloc(edge_num * sizeof(int));
	h_Inum = (int *)malloc(USRNUM * sizeof(int));
	int nownodes = 0;
	for (int i = 0; i < h_adjlist.size(); i++)
	{
		h_head[i] = nownodes;
		h_len[i] = h_adjlist[i].size();
		for (int j = 0; j < h_len[i]; j++)
		{
			h_edges[nownodes] = h_adjlist[i][j];
			nownodes++;
		}
	}

	printf("Bulid network finished\n");

	// set up device
	int dev = 0;
	hipSetDevice(dev);

	// malloc device global memory
	int *d_head, *d_len, *d_edges, *d_Inum;
	hipMalloc((int **)&d_head, USRNUM * sizeof(int));
	hipMalloc((int **)&d_len, USRNUM * sizeof(int));
	hipMalloc((int **)&d_edges, edge_num * sizeof(int));
	hipMalloc((int **)&d_Inum, USRNUM * sizeof(int));

	// transfer data from host to device
	hipMemcpy(d_head, h_head, USRNUM * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_len, h_len, USRNUM * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_edges, h_edges, edge_num * sizeof(int), hipMemcpyHostToDevice);

	//invoke kernel at host side
	dim3 block(1);
	dim3 grid(1);

	SIContagionProcessOnGPU << < grid, block >> > (d_head, d_len, d_edges, d_Inum, USRNUM);

	hipDeviceSynchronize();

	//copy kernel result back to host
	hipMemcpy(h_Inum, d_Inum, USRNUM * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < USRNUM; i++)
	{
		if (h_Inum[i] == 0)
		{
			break;
		}
		printf("%d\n", h_Inum[i]);
	}

	//free
	hipFree(d_head);
	hipFree(d_len);
	hipFree(d_edges);
	hipFree(d_Inum);

	free(h_head);
	free(h_edges);
	free(h_len);
	free(h_Inum);

	system("pause");
	return 0;
}
